#include "hip/hip_runtime.h"
/*
 * Module    : IonSeries
 * Copyright : (c) 2009 Trevor L. McDonell
 * License   : BSD
 */

#include <assert.h>

#include <host_defines.h>
#include <hip/device_functions.h>

#include "cbits/mass.h"
#include "cbits/kernels.h"


/*
 * Convert a given mass into a mass/charge ratio, and locate the appropriate
 * spectrum bin for the peak.
 */
__device__ int
binIonMZ(float mass, int charge)
{
    int bin = rintf((mass + massH * charge) / (charge * binWidthMono));
#ifdef __DEVICE_EMULATION__
    assert(bin >= 0 && bin < 2048 && "index out of bounds");
#endif
    return bin;
}


/*
 * Add a spectral peak for each fragment ion location, as well as the peaks
 * corresponding to the neutral losses of H2O and NH3.
 */
__device__ void
addIonsAB(float mass, int charge, int *spec)
{
    // A
    atomicMax(&spec[binIonMZ(mass - massCO, charge)], 10);

    // B
    int m = binIonMZ(mass, charge);

    atomicMax(&spec[m],   50);
    atomicMax(&spec[m+1], 25);
    atomicMax(&spec[m-1], 25);
    atomicMax(&spec[binIonMZ(mass - massH2O, charge)], 10);
    atomicMax(&spec[binIonMZ(mass - massNH3, charge)], 10);
}


__device__ void
addIonsY(float mass, int charge, int *spec)
{
    int m = binIonMZ(mass + massH2O, charge);

    atomicMax(&spec[m],   50);
    atomicMax(&spec[m+1], 25);
    atomicMax(&spec[m-1], 25);
    atomicMax(&spec[binIonMZ(mass - massNH3, charge)], 10);
}


/*
 * Add a spectral peak for each fragment ion location. The output spectrum array
 * must exist and be initialised to zero.
 */
__global__ void
buildThrySpecXCorr_kernel
(
    int          max_charge,
    float        *b_ions,
    float        *y_ions,
    int          *spec,
    unsigned int len_ions,
    unsigned int len_spec
)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx < len_ions)
    {
        int   charge = 1;
        float b_mass = b_ions[idx];
        float y_mass = y_ions[idx];

        do
        {
            addIonsAB(b_mass, charge, spec);
            addIonsY (y_mass, charge, spec);
        }
        while (++charge < max_charge);
    }
}


/*
 * Kernel wrapper to be called from C. The Ion and Spectrum arrays are in the
 * device memory space.
 */
void
buildThrySpecXCorr
(
    int          charge,
    float        *b_ions,
    float        *y_ions,
    int          *spec,
    unsigned int len_ions,
    unsigned int len_spec
)
{
    int threads = min(len_ions, 64);
    int blocks  = (len_ions + threads - 1) / threads;

    buildThrySpecXCorr_kernel<<<blocks,threads>>>(charge, b_ions, y_ions, spec, len_ions, len_spec);
}

