#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------------------
 *
 * Module    : 
 * Copyright : (c) [2009..2011] Kevin Ying
 * License   : BSD
 *
 * ---------------------------------------------------------------------------*/

#include "utils.h"
#include "device.h"
/*#include "texture.h"*/
/*#include "algorithms.h"*/

#include <stdint.h>
#include <stdlib.h>
#include <iostream>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>
#include "algorithms.h"


/*
 * Scan a warp-sized chunk of data. Because warps execute instructions in SIMD
 * fashion, there is no need to synchronise in order to share data. The most
 * efficient algorithm is the step-efficient method of Hillis & Steele that
 * takes log(N) steps, rather than the work-efficient tree-based algorithm
 * described by Blelloch that takes 2 * log(N) steps.
 */
template <class T, bool inclusive>
static __device__ T
scan_warp(T val, volatile T* s_data)
{
    const uint32_t idx  = threadIdx.x;
    const uint32_t lane = threadIdx.x & (WARP_SIZE-1);

    /*
     * If we double the size of the s_data array and pad the bottom half with
     * zero, then we can avoid branching (although there is plenty already).
     *
     * In device emulation mode, the warp size is 1 and so sync-less operation
     * does not work.
     */
    s_data[idx] = val;                                                        __EMUSYNC;
#ifdef __DEVICE_EMULATION__
    val = (lane >=  1) ? s_data[idx -  1] : 0; __EMUSYNC; s_data[idx] += val; __EMUSYNC;
    val = (lane >=  2) ? s_data[idx -  2] : 0; __EMUSYNC; s_data[idx] += val; __EMUSYNC;
    val = (lane >=  4) ? s_data[idx -  4] : 0; __EMUSYNC; s_data[idx] += val; __EMUSYNC;
    val = (lane >=  8) ? s_data[idx -  8] : 0; __EMUSYNC; s_data[idx] += val; __EMUSYNC;
    val = (lane >= 16) ? s_data[idx - 16] : 0; __EMUSYNC; s_data[idx] += val; __EMUSYNC;
#else
    if (lane >=  1) s_data[idx] = val = val + s_data[idx -  1];
    if (lane >=  2) s_data[idx] = val = val + s_data[idx -  2];
    if (lane >=  4) s_data[idx] = val = val + s_data[idx -  4];
    if (lane >=  8) s_data[idx] = val = val + s_data[idx -  8];
    if (lane >= 16) s_data[idx] = val = val + s_data[idx - 16];
#endif

    if (inclusive) return s_data[idx];
    else           return (lane > 0) ? s_data[idx - 1] : 0;
}

template <uint32_t BlockSize>   
__global__ static void
findModablePeptides_core
(
    uint32_t            *d_valid,

    const uint8_t       *d_ions,
    const uint32_t      *d_tc,
    const uint32_t      *d_tn,

    const uint32_t      *d_sub_idx,
    const uint32_t      sub_idx_length,

    const uint8_t       *d_ma,
    const uint8_t       *d_ma_count,
    const uint32_t      ma_length
)
{

    assert(BlockSize % WARP_SIZE == 0);

    const uint32_t vectorsPerBlock = BlockSize / WARP_SIZE;
    const uint32_t numVectors      = vectorsPerBlock * gridDim.x;
    const uint32_t thread_id       = BlockSize * blockIdx.x + threadIdx.x;
    const uint32_t vector_id       = thread_id / WARP_SIZE;
    const uint32_t thread_lane     = threadIdx.x & (WARP_SIZE-1);

    __shared__ volatile uint32_t s_data[BlockSize];

    for (uint32_t row = vector_id; row < sub_idx_length; row += numVectors)
    {
        const uint32_t idx       = d_sub_idx[row];
        const uint32_t row_start = d_tc[idx];
        const uint32_t row_end   = d_tn[idx];

        s_data[threadIdx.x] = 0;


        for (uint32_t j = row_start + thread_lane; j < row_end; j += WARP_SIZE)
        {
            // check if acid is one that should be modded
            uint32_t count = 0;
            if (d_ma[0] == d_ions[j])
                count++;

            if (thread_lane == 0)
                count += s_data[threadIdx.x + (WARP_SIZE-1)];
            
            count = scan_warp<uint32_t, true>(count, s_data); 

            if (j == row_end-1) {
                d_valid[row] = count;
            }

        }

    }
}

/*
 * Select a number of threads and blocks. Each block will have at least one full
 * warp, as required by the core kernel
 */
static void
findByMod_control(uint32_t N, uint32_t &blocks, uint32_t &threads)
{
    threads = (N < MAX_THREADS) ? max(WARP_SIZE, ceilPow2(N)) : MAX_THREADS;
    blocks  = (N + threads - 1) / threads;
    blocks  = min(blocks, MAX_BLOCKS);
}

template <typename T>
struct greaterThan : public thrust::unary_function<T,bool>
{
    T bound;

    __host__ __device__
    greaterThan(T _m) : bound(_m) {}

    __host__ __device__ bool operator() (T x)
    {
        return (bound < x);
    }
};

uint32_t
findModablePeptides
(
    uint32_t            *d_out_raw,

    const uint8_t       *d_ions,
    const uint32_t      *d_tc,
    const uint32_t      *d_tn,

    const uint32_t      *d_sub_idx,
    const uint32_t      sub_idx_length,

    const uint8_t       *d_ma,
    const uint8_t       *d_ma_count,
    const uint32_t      ma_length
)
{
    printf("This is findModablePeptides\n");
    uint32_t            threads;
    uint32_t            blocks;
    uint32_t            *d_valid_raw;

    CUDA_SAFE_CALL( hipMalloc((void**) &d_valid_raw, sub_idx_length* sizeof(uint32_t)) );

    // control
    findByMod_control(sub_idx_length, blocks, threads);

    // core
    switch (threads)
    {
    case 128: findModablePeptides_core<128><<<blocks, threads>>>(d_valid_raw, d_ions, d_tc, d_tn, d_sub_idx, sub_idx_length, d_ma, d_ma_count, ma_length); break;
    case  64: findModablePeptides_core< 64><<<blocks, threads>>>(d_valid_raw, d_ions, d_tc, d_tn, d_sub_idx, sub_idx_length, d_ma, d_ma_count, ma_length); break;
    case  32: findModablePeptides_core< 32><<<blocks, threads>>>(d_valid_raw, d_ions, d_tc, d_tn, d_sub_idx, sub_idx_length, d_ma, d_ma_count, ma_length); break;
    }

    // compact
    /*uint32_t N;*/
    /*N = compactIndices((uint32_t*)  d_out, d_valid, sub_idx_length);*/

    thrust::device_ptr<const uint32_t>  d_valid(d_valid_raw);
    thrust::device_ptr<uint32_t>        d_out(d_out_raw);

    // print before compaction
    thrust::host_vector<uint32_t> H(d_valid, d_valid + sub_idx_length);
    std::cout << "Printing results. no. in subset " << sub_idx_length << std::endl;
    for(int i = 0; i < sub_idx_length; i++) {
        std::cout << "d_valid " << H[i] << std::endl;
    }
   
    // copy if there are enough aa's to apply the mod
    thrust::device_ptr<const uint32_t> d_sub_idx_th(d_sub_idx); 
    thrust::device_ptr<uint32_t> d_out_end =
        thrust::copy_if(d_sub_idx_th, d_sub_idx_th + sub_idx_length, d_valid, d_out, greaterThan<const uint32_t>(0));


    hipFree(d_valid_raw);
    /*hipMemcpy(&h_total, (d_rtotal + num_nIdx -1), sizeof(uint32_t), hipMemcpyDeviceToHost);*/


    // print d_out after compaction
    thrust::host_vector<uint32_t> H_compact(d_out, d_out_end);
    std::cout << "Printing results" << std::endl;
    for(int i = 0; i < d_out_end - d_out; i++) {
        std::cout << "d_out " << H_compact[i] << std::endl;
    }
    

    return 0;
}

