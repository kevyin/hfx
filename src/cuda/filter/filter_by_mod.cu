/* -----------------------------------------------------------------------------
 *
 * Module    : 
 * Copyright : (c) [2009..2011] Kevin Ying
 * License   : BSD
 *
 * ---------------------------------------------------------------------------*/

#include "utils.h"
#include "device.h"
#include "algorithms.h"
#include "functors.h"
#include "functional.hpp"

#include <stdint.h>
#include <stdlib.h>
#include <iostream>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/transform_scan.h>
#include <thrust/binary_search.h>
#include <thrust/sequence.h>
#include <thrust/remove.h>
#include <thrust/copy.h>
#include <thrust/scan.h>

#include <time.h>
#include <stdio.h>

using namespace thrust;

/**
 * For each modification, finds peptides within range by finding the begin and end ranges to pep_idx_r_sorted
 */
uint32_t
findBeginEnd_f
(
    uint32_t            *d_begin_raw,
    uint32_t            *d_end_raw,
    //uint32_t            *d_num_pep_raw,
    uint32_t            *d_num_pep_scan_raw,

    const float         *d_r,
    const uint32_t      *d_pep_idx_r_sorted,
    const uint32_t      num_pep,

    const float         *d_mod_delta_raw,
    const uint32_t      num_mod,
    const float         mass,
    const float         eps
)
{
#ifdef _BENCH
    hipDeviceSynchronize();
    time_t t_beg, t_end;
    time(&t_beg);
#endif 

    if (num_mod == 0) return 0;
    // setup device ptrs
    thrust::device_ptr<uint32_t> d_begin(d_begin_raw);
    thrust::device_ptr<uint32_t> d_end(d_end_raw);
    //thrust::device_ptr<uint32_t> d_num_pep(d_num_pep_raw);
    thrust::device_ptr<uint32_t> d_num_pep_scan(d_num_pep_scan_raw);
    thrust::device_ptr<const float> d_mod_delta(d_mod_delta_raw);
    device_ptr<const float> d_r_th(d_r);
    device_ptr<const uint32_t> d_pep_idx_r_sorted_th(d_pep_idx_r_sorted);

    // Find begin and ends

    permutation_iterator<device_ptr<const float>, device_ptr<const uint32_t> > iter(d_r_th, d_pep_idx_r_sorted_th);
    lower_bound(iter,iter+num_pep,
                make_transform_iterator(d_mod_delta, thrust::bind1st(minus<float>(), mass-eps)),
                make_transform_iterator(d_mod_delta+num_mod, thrust::bind1st(minus<float>(), mass-eps)),
                d_begin);

    upper_bound(iter,iter+num_pep,
                make_transform_iterator(d_mod_delta, thrust::bind1st(minus<float>(), mass+eps)),
                make_transform_iterator(d_mod_delta+num_mod, thrust::bind1st(minus<float>(), mass+eps)),
                d_end);

    // calc number of peptides in a range and scan values

    thrust::transform_exclusive_scan(
        thrust::make_zip_iterator(thrust::make_tuple(d_end, d_begin)),
        thrust::make_zip_iterator(thrust::make_tuple(d_end + num_mod, d_begin + num_mod)),
        d_num_pep_scan,
        minus_2tuple<uint32_t>(),
        0,
        thrust::plus<uint32_t>()); 
    

#ifdef _DEBUG
// check begin and ends are sane
    thrust::device_ptr<const float> d_r_th(d_r);
    for (uint32_t i = 0; i < num_mod; ++i) {
        const float target_l = mass - d_mod_delta[i] - eps;
        const float target_u = mass - d_mod_delta[i] + eps;

        //std::cerr << "num_pep " << d_num_pep[i] << " scan " << d_num_pep_scan[i] << " beg " << d_begin[i] << " end " << d_end[i] << std::endl;
        if (d_begin[i] < d_end[i]) {
            const float beginMass = d_r_th[d_pep_idx_r_sorted_th[d_begin[i]]];
            const float lastMass = d_r_th[d_pep_idx_r_sorted_th[d_end[i] - 1]];

            //std::cout << target_l << " " << beginMass << " " << d_begin[i] << " " << d_num_pep[i] << " " << d_end[i] << " " << lastMass << " " << target_u << std::endl;
            if (!(target_l <= beginMass) ||
                !(target_u >= lastMass)) {
                std::cerr << "findBeginEnd doesn't seem to be correct (1)" << std::endl;
                exit(1);
            }
            if (d_begin[i] > 0) {
                const float beginMass_ = d_r_th[d_pep_idx_r_sorted_th[d_begin[i] - 1]];
                if (!(target_l > beginMass_)) {
                    std::cerr << "findBeginEnd doesn't seem to be correct (2)" << std::endl;
                    exit(2);
                }
            }
            if (d_end[i] < num_pep) {
                const float endMass_ = d_r_th[d_pep_idx_r_sorted_th[d_end[i]]];
                if (!(target_u < endMass_)) {
                    std::cerr << "findBeginEnd doesn't seem to be correct (3)" << std::endl;
                    exit(3);
                }
            }
        } else if (d_begin[i] > d_end[i]) {
                    std::cerr << "findBeginEnd doesn't seem to be correct (4)" << std::endl;
                    exit(4);

        }
    }
#endif

    //uint32_t total = thrust::reduce(d_num_pep, d_num_pep + num_mod);

#ifdef _BENCH
    hipDeviceSynchronize();
    time(&t_end);
    std::cerr << "Time elapsed for findBeginEnd_f: " << difftime(t_end,t_beg) << " %.2lf seconds\n" << std::endl;
#endif 
    // return total peptides
    uint32_t last_mod_idx = num_mod - 1;
    return d_num_pep_scan[last_mod_idx] + d_end[last_mod_idx] - d_begin[last_mod_idx];
}

/**
 * The peptides from begin to end are laid out to be search in parallel together
 * pep_idx is the idx to the original r,c,n array
 * pep_mod_idx is the idx to the modification which it can be applied to
 */
template <typename T>
struct fillPepAndModIdx : public thrust::unary_function<T, void>
{
    //thrust::device_ptr<T> d_out_pep_idx;
    T *d_out_pep_idx;
    T *d_out_pep_mod_idx;

    const T *d_pep_idx_r_sorted;
    const T *d_begin;
    const T *d_end;
    const T *d_num_pep_scan;
    const T *d_spec_num_cand_scan;
    const T num_mod;

    __host__ __device__
    fillPepAndModIdx(T *_pi,
                     T *_pmi,
                     const T *_pirs,
                     const T *_b,
                     const T *_e,
                     const T *_nps,
                     const T *_sncs,
                     const T _nm)
                    : d_out_pep_idx(_pi), d_out_pep_mod_idx(_pmi), d_pep_idx_r_sorted(_pirs), d_begin(_b), d_end(_e), d_num_pep_scan(_nps), d_spec_num_cand_scan(_sncs), num_mod(_nm) {}

    __host__ __device__ void operator() (T idx)
    {
        const uint32_t mod_idx = idx % num_mod;
        const uint32_t spec_idx = idx / num_mod;
        const size_t offset = (spec_idx * num_mod) + mod_idx;
        T pep_idx;
        T out_idx = d_num_pep_scan[offset] + d_spec_num_cand_scan[spec_idx]; // position in the out array
        for (T i = d_begin[offset]; i != d_end[offset]; ++i) {
            pep_idx = d_pep_idx_r_sorted[i];

            d_out_pep_idx[out_idx] = pep_idx;
            d_out_pep_mod_idx[out_idx] = mod_idx;
            ++out_idx;
        }
    }
};

template <typename T>
struct fillPepMACount: public thrust::unary_function<T, T>
{
    const T *d_pep_idx;
    const T *d_tc;
    const T *d_tn;
    const uint8_t *d_ions;
    const uint8_t *d_ma;
    const T num_ma;  

    __host__ __device__
    fillPepMACount(const T *_pep_idx,
                   const T *_tc,
                   const T *_tn,
                   const uint8_t *_ions,
                   const uint8_t *_ma,
                   const T _num_ma)
                    : d_pep_idx(_pep_idx),
                      d_tc(_tc),
                      d_tn(_tn),
                      d_ions(_ions),
                      d_ma(_ma),
                      num_ma(_num_ma) {}

    __host__ __device__ T operator() (T i)
    {
        T idx     = i / num_ma;
        T pep_idx = d_pep_idx[idx];
        T ma_idx  = i % num_ma;
        uint8_t ma  = d_ma[ma_idx];
        T count   = 0;
        for (T a = d_tc[pep_idx] ; a < d_tn[pep_idx]; ++a) {
            uint8_t ion = GET_ACID_CHAR(d_ions[a]);    
            if (ion == ma) {
                count++;
            }
        }
        return count;
    }
};

template <typename T>
struct checkModable: public thrust::unary_function<T, T>
{
    const T *d_pep_ma_count;
    const T *d_pep_mod_idx;
    const T *d_mod_ma_count;
    const T num_ma;  

    __host__ __device__
    checkModable(const T *_pep_ma_count,
                 const T *_pep_mod_idx,
                 const T *_mod_ma_count,
                 const T _num_ma)
                 : d_pep_ma_count(_pep_ma_count),
                   d_pep_mod_idx(_pep_mod_idx),
                   d_mod_ma_count(_mod_ma_count),
                   num_ma(_num_ma) {}

    __host__ __device__ T operator() (T idx)
    {
        T pep_start = num_ma * idx ;
        T mod_start = num_ma * d_pep_mod_idx[idx];
        for (T ma = 0; ma < num_ma; ++ma) {
            if (d_pep_ma_count[pep_start + ma] < d_mod_ma_count[mod_start + ma]) {
                return false;
            }
        }
        return true;
    }
};

// Using thrust
uint32_t
findModablePeptides
(
    uint32_t            *d_out_pep_idx_raw, 
    uint32_t            *d_out_pep_mod_idx_raw, 
    uint32_t            *d_out_pep_ma_count_raw,   // 2d array, count of each ma in each peptide
    uint32_t            *d_out_spec_num_valid_scan, 

    const uint32_t      *d_spec_num_cand_scan_raw, // containing num cand by mass
    const uint32_t      num_spec,
    uint32_t            num_cand_total,

    const uint8_t       *d_ions,
    const uint32_t      *d_tc,
    const uint32_t      *d_tn,

    const uint32_t      *d_pep_idx_r_sorted_raw,

    const uint32_t      *d_begin_raw,
    const uint32_t      *d_end_raw,
    const uint32_t      *d_num_pep_scan_raw,
    const uint32_t      *d_mod_ma_count,
    const uint32_t      num_mod,

    const uint8_t       *d_ma,
    const uint32_t      num_ma
)
{
#ifdef _BENCH
    hipDeviceSynchronize();
    std::cerr<< "findModablePeptides" << std::endl;
    time_t t_beg, t_end;
    time(&t_beg);
    //printGPUMemoryUsage();
#endif 

    thrust::device_ptr<uint32_t> d_out_pep_ma_count(d_out_pep_ma_count_raw);

    thrust::device_ptr<const uint32_t> d_spec_num_cand_scan(d_spec_num_cand_scan_raw);

    // fill arrays mod_idx and pep_idx. will look like:
    // mod_idx  pep_idx
    // 0        a0
    // 0        .
    // 0        .
    // 0        b0
    // 1        a1
    // 1        .
    // 1        .
    // 1        b1
    // Where a{n} .. b{n} are indices to the peptides which are inrange for the modificiation 
    thrust::counting_iterator<uint32_t> first(0);
    thrust::counting_iterator<uint32_t> last = first + num_mod*num_spec;
    thrust::for_each(first, last, fillPepAndModIdx<uint32_t>(d_out_pep_idx_raw, d_out_pep_mod_idx_raw, d_pep_idx_r_sorted_raw, d_begin_raw, d_end_raw, d_num_pep_scan_raw, d_spec_num_cand_scan.get(), num_mod));

    // non compacted arrays
    thrust::device_vector<bool> d_valid_v(num_cand_total);
    /*thrust::device_vector<uint32_t> d_out_pep_ma_count2(num_ma*num_cand_total);*/

    // fill counts and check if modable
    
    last = first + num_cand_total * num_ma;
    thrust::transform(first, last, d_out_pep_ma_count, fillPepMACount<uint32_t>(d_out_pep_idx_raw, d_tc, d_tn, d_ions, d_ma, num_ma));

    last = first + num_cand_total;
    thrust::transform(first, last, d_valid_v.begin(), checkModable<uint32_t>(d_out_pep_ma_count_raw, d_out_pep_mod_idx_raw, d_mod_ma_count, num_ma));

    // compact
    device_ptr<uint32_t> d_pep_mod_idx(d_out_pep_mod_idx_raw);
    device_ptr<uint32_t> d_pep_idx(d_out_pep_idx_raw);
    device_ptr<uint32_t> d_pep_ma_count(d_out_pep_ma_count_raw);

    device_vector<uint32_t> d_spec_cand_idx(num_cand_total);
    sequence(d_spec_cand_idx.begin(),d_spec_cand_idx.end());

    typedef device_ptr<uint32_t>            UIntDIter;
    typedef device_vector<uint32_t>::iterator DVUIntDIter;
    typedef tuple<UIntDIter, UIntDIter, DVUIntDIter>     UIntDIterTuple3;
    typedef zip_iterator<UIntDIterTuple3>   ZipIter;

    ZipIter end = remove_if(
        make_zip_iterator(make_tuple(d_pep_mod_idx, 
                                     d_pep_idx, 
                                     d_spec_cand_idx.begin())),
        make_zip_iterator(make_tuple(d_pep_mod_idx+num_cand_total, 
                                     d_pep_idx+num_cand_total, 
                                     d_spec_cand_idx.end())),
        d_valid_v.begin(),
        logical_not<bool>());

    last = first + num_cand_total * num_ma;
    remove_if(d_pep_ma_count, d_pep_ma_count+num_cand_total*num_ma,
              make_transform_iterator(first, mat_wider<bool>(d_valid_v.data().get(),num_ma)),
              logical_not<bool>());
    
    UIntDIterTuple3 endTuple = end.get_iterator_tuple();
    const uint32_t numValid = get<0>(endTuple) - d_pep_mod_idx;

    // now calc num valid per spectrum to differentiate the sections
    thrust::lower_bound(d_spec_cand_idx.begin(), get<2>(endTuple),
                        d_spec_num_cand_scan, d_spec_num_cand_scan + num_spec,
                        device_ptr<uint32_t>(d_out_spec_num_valid_scan));




#ifdef _BENCH
    hipDeviceSynchronize();
    time(&t_end);
    std::cerr << "Time elapsed for findModablePeptides: " << difftime(t_end,t_beg) << "%.2lf seconds\n" << std::endl; 
#endif 

    return numValid;
}
