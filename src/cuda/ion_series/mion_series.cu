#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------------------
 *
 * Module    : Ion Series
 * Copyright : (c) [2009..2011] Kevin Ying
 * License   : BSD
 *
 * ---------------------------------------------------------------------------*/

#include "utils.h"
#include "device.h"
#include "texture.h"
#include "ion_series.h"
#include "algorithms.h"

#include <stdint.h>


/*
 * Scan a warp-sized chunk of data. Because warps execute instructions in SIMD
 * fashion, there is no need to synchronise in order to share data. The most
 * efficient algorithm is the step-efficient method of Hillis & Steele that
 * takes log(N) steps, rather than the work-efficient tree-based algorithm
 * described by Blelloch that takes 2 * log(N) steps.
 */
template <class T, bool inclusive>
static __device__ T
scan_warp(T val, volatile T* s_data)
{
    const uint32_t idx  = threadIdx.x;
    const uint32_t lane = threadIdx.x & (WARP_SIZE-1);

    /*
     * If we double the size of the s_data array and pad the bottom half with
     * zero, then we can avoid branching (although there is plenty already).
     *
     * In device emulation mode, the warp size is 1 and so sync-less operation
     * does not work.
     */
    s_data[idx] = val;                                                        __EMUSYNC;
#ifdef __DEVICE_EMULATION__
    val = (lane >=  1) ? s_data[idx -  1] : 0; __EMUSYNC; s_data[idx] += val; __EMUSYNC;
    val = (lane >=  2) ? s_data[idx -  2] : 0; __EMUSYNC; s_data[idx] += val; __EMUSYNC;
    val = (lane >=  4) ? s_data[idx -  4] : 0; __EMUSYNC; s_data[idx] += val; __EMUSYNC;
    val = (lane >=  8) ? s_data[idx -  8] : 0; __EMUSYNC; s_data[idx] += val; __EMUSYNC;
    val = (lane >= 16) ? s_data[idx - 16] : 0; __EMUSYNC; s_data[idx] += val; __EMUSYNC;
#else
    if (lane >=  1) s_data[idx] = val = val + s_data[idx -  1];
    if (lane >=  2) s_data[idx] = val = val + s_data[idx -  2];
    if (lane >=  4) s_data[idx] = val = val + s_data[idx -  4];
    if (lane >=  8) s_data[idx] = val = val + s_data[idx -  8];
    if (lane >= 16) s_data[idx] = val = val + s_data[idx - 16];
#endif

    if (inclusive) return s_data[idx];
    else           return (lane > 0) ? s_data[idx - 1] : 0;
}


__inline__ __device__ static float
ionMZ(const float m, const float c)
{
    return __fdividef(m + MASS_H * c, c);
}

__inline__ __device__ static uint32_t
binMZ(const float mz)
{
    return rintf(__fdividef(mz, BIN_WIDTH_MONO));
}

__inline__ __device__ static void
addIon(uint32_t *d_spec, const uint32_t N, const int32_t x, const uint32_t y)
{
    if (0 <= x && x < N) atomicMax(&d_spec[x], y);
}


template <uint32_t charge>
__device__ void
addIonsAB(uint32_t *d_spec, const uint32_t N, const float mass)
{
    float   m;
    int32_t x;

    // A-ions
    addIon(d_spec, N, binMZ(ionMZ(mass - MASS_CO, charge)), 10);

    // B-ions
    m = ionMZ(mass, charge);
    x = binMZ(m);

    addIon(d_spec, N, x,   50);
    addIon(d_spec, N, x+1, 25); // technically, should be binMZ(m+1)
    addIon(d_spec, N, x-1, 25);

    addIon(d_spec, N, binMZ(m - __fdividef(MASS_H2O, charge)), 10);
    addIon(d_spec, N, binMZ(m - __fdividef(MASS_NH3, charge)), 10);
}


template <uint32_t charge>
__device__ void
addIonsY(uint32_t *d_spec, const uint32_t N, const float mass)
{
    float   m = ionMZ(mass + MASS_H2O, charge);
    int32_t x = binMZ(m);

    // Y-ions
    addIon(d_spec, N, x,   50);
    addIon(d_spec, N, x+1, 25);
    addIon(d_spec, N, x-1, 25);

    addIon(d_spec, N, binMZ(m - __fdividef(MASS_NH3, charge)), 10);
}


template <uint32_t charge>
__device__ void
addIons_k(uint32_t *d_spec, const uint32_t N, const float b_mass, const float y_mass)
{
    addIonsAB<charge>(d_spec, N, b_mass);
    addIonsY <charge>(d_spec, N, y_mass);
}


/*
 * Return the mass of an amino acid residue in atomic mass units, for the given
 * short abbreviation.
 */
template <bool UseCache>
__device__ float
getAAMass(const float *d_mass, const char aa)
{
    return fetch_x<UseCache>(aa - 'A', d_mass);
}

/*
 * Determine how much the modification shift the residual mass by
 * @TODO passing this by value would save computing this every time
 */
template <bool UseCache>
__device__ float
getModResOffset(const float *d_mass, const uint8_t *d_ma, const uint8_t *d_ma_count, const uint32_t num_ma) 
{
    float offset = 0;
    for (uint32_t i = 0; i < num_ma; ++i)
    {
        for (uint8_t j = 0; j < d_ma_count[i]; ++j)
        {
            offset += getAAMass<UseCache>(d_mass, d_ma[i]);
        }
    }
    return offset;
}


/*
 * Generate theoretical spectra for a collection of peptide fragments. The
 * 'ions' array contains the individual amino-acid masses for the database
 * entries. We are interested in the sequences generated between the terminal
 * indices (tc,tn) of the locations specified in the 'idx' array.
 *
 * A warp of threads iterates between the (tc,tn) indices, generating the b- and
 * y-ion mass ladders. A (long) sequence of (slow) global atomic update requests
 * is subsequently issued. The input d_spec should be initially zero, and on
 * output will contain the theoretical spectra peaks in a dense (although
 * mostly zero) matrix.
 */
template <uint32_t BlockSize, uint32_t MaxCharge, bool UseCache>
__global__ static void
addModIons_core
(
    uint32_t            *d_mspec,
    const float         *d_residual,    // peptide residual mass
    const float         *d_mass,        // lookup table for ion character codes ['A'..'Z']
    const uint8_t       *d_ions,        // individual ion character codes (the database)
    const uint32_t      *d_tc,          // c-terminal indices
    const uint32_t      *d_tn,          // n-terminal indices
    const uint32_t      *d_mpep_idx,
    const uint32_t      *d_mpep_mcomb,
    const uint32_t      num_mpep,
    const uint8_t       *d_ma,
    const uint8_t       *d_ma_count,
    const uint32_t      num_ma,
    const uint32_t      len_spec
)
{
    assert(BlockSize % WARP_SIZE == 0);

    const uint32_t vectorsPerBlock = BlockSize / WARP_SIZE;
    const uint32_t numVectors      = vectorsPerBlock * gridDim.x;
    const uint32_t thread_id       = BlockSize * blockIdx.x + threadIdx.x;
    const uint32_t vector_id       = thread_id / WARP_SIZE;
    const uint32_t thread_lane     = threadIdx.x & (WARP_SIZE-1);

    __shared__ volatile float s_data[BlockSize];
    // Keep a record of ith moddable acid as the pep is traversed
    __shared__ volatile uint32_t s_pep_ith_ma[MAX_MA][BlockSize];


    for (uint32_t row = vector_id; row < num_mpep; row += numVectors)
    {
        const uint32_t mpep_idx       = d_mpep_idx[row];
        const uint32_t row_start = d_tc[mpep_idx];
        const uint32_t row_end   = d_tn[mpep_idx];
        const float    residual  = d_residual[mpep_idx] + getModResOffset<UseCache>(d_mass, d_ma, d_ma_count, num_ma);

        uint32_t       *spec     = &d_mspec[row * len_spec];
        float          b_mass;
        float          y_mass;

        s_data[threadIdx.x]      = 0;
        for (int mod = 0; mod < num_ma; mod++)
        {
            s_pep_ith_ma[mod][threadIdx.x] = 0;
        }

        /*
         * Have all threads read in values for this segment, writing the
         * spectral peaks out to global memory (very, very slowly...)
         */
        for (uint32_t j = row_start + thread_lane; j < row_end; j += WARP_SIZE)
        {
            /*
             * Load the ion mass, and propagate the partial scan results
             */
            // is this acid modable
            bool is_ma= false;
            for (int mod = 0; mod < num_ma && !is_ma; mod++) 
            {
                uint32_t count = 0;
                if (d_ma[mod] == d_ions[j]) 
                {
                    is_ma = true;
                    count++;
                }

                if (thread_lane == 0)
                    count += s_pep_ith_ma[mod][threadIdx.x + (WARP_SIZE-1)];
                
                count = scan_warp<uint32_t, true>(count, s_pep_ith_ma[mod]); 
            }

            if (is_ma) //&& /* is modable according to ith comb*/)
            {
                //b_mass = getModifiedAAMass<UseCache>(d_ma_mass, d_ions[j]);
                b_mass = 1234567;
            } else {
                b_mass = getAAMass<UseCache>(d_mass, d_ions[j]);
            }

            if (thread_lane == 0)
            {
                b_mass += s_data[threadIdx.x + (WARP_SIZE-1)];
            }

            /*
             * Generate fragment mass ladder
             */
            b_mass = scan_warp<float,true>(b_mass, s_data);
            y_mass = residual - b_mass;

            if (1 <= MaxCharge) addIons_k<1>(spec, len_spec, b_mass, y_mass);
            if (2 <= MaxCharge) addIons_k<2>(spec, len_spec, b_mass, y_mass);
            if (3 <= MaxCharge) addIons_k<3>(spec, len_spec, b_mass, y_mass);
            if (4 <= MaxCharge) addIons_k<4>(spec, len_spec, b_mass, y_mass);
        }
    }
}


/*
 * Select a number of threads and blocks. Each block will have at least one full
 * warp, as required by the core kernel
 */
static void
addModIons_control(uint32_t N, uint32_t &blocks, uint32_t &threads)
{
    threads = (N < MAX_THREADS) ? max(WARP_SIZE, ceilPow2(N)) : MAX_THREADS;
    blocks  = (N + threads - 1) / threads;
    blocks  = min(blocks, MAX_BLOCKS);
}


template <uint32_t MaxCharge, bool UseCache>
static void
addModIons_dispatch
(
    uint32_t            *d_mspec,
    const float         *d_residual,
    const float         *d_mass,
    const uint8_t       *d_ions,
    const uint32_t      *d_tc,
    const uint32_t      *d_tn,
    const uint32_t      *d_mpep_idx,
    const uint32_t      *d_mpep_mcomb,
    const uint32_t      num_mpep,
    const uint8_t       *d_ma,
    const uint8_t       *d_ma_count,
    const uint32_t      num_ma,
    const uint32_t      len_spec
)
{
    uint32_t blocks;
    uint32_t threads;

    if (UseCache)
        bind_x(d_mass);

    addModIons_control(num_mpep, blocks, threads);
    switch (threads)
    {
    //case 512: addModIons_core<512,MaxCharge,UseCache><<<blocks,threads>>>(d_mspec, d_residual, d_mass, d_ions, d_tc, d_tn, d_mpep_idx,, d_mpep_mcomb, num_mpep, d_ma, d_ma_count, len_spec); break;
    //case 256: addModIons_core<256,MaxCharge,UseCache><<<blocks,threads>>>(d_mspec, d_residual, d_mass, d_ions, d_tc, d_tn, d_mpep_idx,, d_mpep_mcomb, num_mpep, d_ma, d_ma_count, len_spec); break;
    case 128: addModIons_core<128,MaxCharge,UseCache><<<blocks,threads>>>(d_mspec, d_residual, d_mass, d_ions, d_tc, d_tn, d_mpep_idx, d_mpep_mcomb, num_mpep, d_ma, d_ma_count, num_ma, len_spec); break;
    case  64: addModIons_core< 64,MaxCharge,UseCache><<<blocks,threads>>>(d_mspec, d_residual, d_mass, d_ions, d_tc, d_tn, d_mpep_idx, d_mpep_mcomb, num_mpep, d_ma, d_ma_count, num_ma, len_spec); break;
    case  32: addModIons_core< 32,MaxCharge,UseCache><<<blocks,threads>>>(d_mspec, d_residual, d_mass, d_ions, d_tc, d_tn, d_mpep_idx, d_mpep_mcomb, num_mpep, d_ma, d_ma_count, num_ma, len_spec); break;
    default:
        assert(!"Non-exhaustive patterns in match");
    }

    if (UseCache)
      unbind_x(d_mass);
}

void addModIons
(
    uint32_t            *d_mspec,
    const float         *d_residual,
    const float         *d_mass,
    const uint8_t       *d_ions,
    const uint32_t      *d_tc,
    const uint32_t      *d_tn,
    const uint32_t      *d_mpep_idx,
    const uint32_t      *d_mpep_mcomb,
    const uint32_t      num_mpep,
    const uint8_t       *d_ma,
    const uint8_t       *d_ma_count,
    const uint32_t      num_ma,
    const uint32_t      max_charge,
    const uint32_t      len_spec
)
{
    switch (max_charge)
    {
    case 1: addModIons_dispatch<1,true>(d_mspec, d_residual, d_mass, d_ions, d_tc, d_tn, d_mpep_idx, d_mpep_mcomb, num_mpep, d_ma, d_ma_count, num_ma, len_spec); break;
    case 2: addModIons_dispatch<2,true>(d_mspec, d_residual, d_mass, d_ions, d_tc, d_tn, d_mpep_idx, d_mpep_mcomb, num_mpep, d_ma, d_ma_count, num_ma, len_spec); break;
    case 3: addModIons_dispatch<3,true>(d_mspec, d_residual, d_mass, d_ions, d_tc, d_tn, d_mpep_idx, d_mpep_mcomb, num_mpep, d_ma, d_ma_count, num_ma, len_spec); break;
    case 4: addModIons_dispatch<4,true>(d_mspec, d_residual, d_mass, d_ions, d_tc, d_tn, d_mpep_idx, d_mpep_mcomb, num_mpep, d_ma, d_ma_count, num_ma, len_spec); break;
    default:
        assert(!"Non-exhaustive patterns in match");
    }
}

