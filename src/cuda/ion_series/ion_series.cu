#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------------------
 *
 * Module    : Ion Series
 * Copyright : (c) 2009 Trevor L. McDonell
 * License   : BSD
 *
 * ---------------------------------------------------------------------------*/

#include "utils.h"
#include "ion_series.h"
#include "algorithms.h"

#include <stdint.h>


__inline__ __device__ static float
ionMZ(const float m, const float c)
{
    return __fdividef(m + MASS_H * c, c);
}

__inline__ __device__ static uint32_t
binMZ(const float mz)
{
    return rintf(__fdividef(mz, BIN_WIDTH_MONO));
}

__inline__ __device__ static void
addIon(uint32_t *d_spec, const uint32_t N, const int32_t x, const uint32_t y)
{
    if (0 <= x && x < N) atomicMax(&d_spec[x], y);
}


template <uint32_t charge>
__device__ void
addIonsAB(uint32_t *d_spec, const uint32_t N, const float mass)
{
    float   m;
    int32_t x;

    // A-ions
    addIon(d_spec, N, binMZ(ionMZ(mass - MASS_CO, charge)), 10);

    // B-ions
    m = ionMZ(mass, charge);
    x = binMZ(m);

    addIon(d_spec, N, x,   50);
    addIon(d_spec, N, x+1, 25); // technically, should be binMZ(m+1)
    addIon(d_spec, N, x-1, 25);

    addIon(d_spec, N, binMZ(m - __fdividef(MASS_H2O, charge)), 10);
    addIon(d_spec, N, binMZ(m - __fdividef(MASS_NH3, charge)), 10);
}


template <uint32_t charge>
__device__ void
addIonsY(uint32_t *d_spec, const uint32_t N, const float mass)
{
    float   m = ionMZ(mass + MASS_H2O, charge);
    int32_t x = binMZ(m);

    // Y-ions
    addIon(d_spec, N, x,   50);
    addIon(d_spec, N, x+1, 25);
    addIon(d_spec, N, x-1, 25);

    addIon(d_spec, N, binMZ(m - __fdividef(MASS_NH3, charge)), 10);
}


template <uint32_t charge>
__device__ void
addIons_k(uint32_t *d_spec, const uint32_t N, const float b_mass, const float y_mass)
{
    addIonsAB<charge>(d_spec, N, b_mass);
    addIonsY <charge>(d_spec, N, y_mass);
}


/*
 * Generate theoretical spectra for a collection of peptide fragments. The
 * yIonLadder array contains data for all fragments in the database, although we
 * are only interested in those beginning at the inRangeIdx positions.
 *
 * A warp of threads iterates over the fragment masses for a peptide, issuing a
 * (long) sequence of (slow) global atomic update requests. The input spectra
 * matrix should be initially zero, and on output will contain the theoretical
 * spectral peaks in a square (but mostly sparse) matrix.
 */
template <uint32_t BlockSize, uint32_t MaxCharge>
__global__ static void
addIons_core
(
    uint32_t            *d_spec,
    const float         *d_residual,
    const float         *d_yIonLadder,
    const uint32_t      *d_rowPtr,
    const uint32_t      *d_inRangeIdx,
    const uint32_t      num_inRange,
    const uint32_t      len_spec
)
{
    assert(BlockSize % WARP_SIZE == 0);

    const uint32_t vectorsPerBlock = BlockSize / WARP_SIZE;
    const uint32_t numVectors      = vectorsPerBlock * gridDim.x;
    const uint32_t thread_id       = BlockSize * blockIdx.x + threadIdx.x;
    const uint32_t vector_id       = thread_id / WARP_SIZE;
    const uint32_t thread_lane     = threadIdx.x & (WARP_SIZE-1);
    const uint32_t vector_lane     = threadIdx.x / WARP_SIZE;

    __shared__ volatile uint32_t s_ptrs[vectorsPerBlock][2];

    for (uint32_t row = vector_id; row < num_inRange; row += numVectors)
    {
        const uint32_t idx      = d_inRangeIdx[row];
        const float    residual = d_residual[idx];
        uint32_t       *spec    = &d_spec[row * len_spec];

        /*
         * Use two threads to fetch the indices of the start and end of this
         * segment. This is a single coalesced (unaligned) global read.
         */
        if (thread_lane < 2)
            s_ptrs[vector_lane][thread_lane] = d_rowPtr[idx + thread_lane];

        __EMUSYNC;
        const uint32_t row_start = s_ptrs[vector_lane][0];
        const uint32_t row_end   = s_ptrs[vector_lane][1];

        /*
         * Have all threads read in values for this segment, writing the
         * spectral peaks out to global memory (very, very slowly...)
         */
        for (uint32_t j = row_start + thread_lane; j < row_end; j += WARP_SIZE)
        {
            const float y_mass = d_yIonLadder[j];
            const float b_mass = residual - y_mass;

            if (1 <= MaxCharge) addIons_k<1>(spec, len_spec, b_mass, y_mass);
            if (2 <= MaxCharge) addIons_k<2>(spec, len_spec, b_mass, y_mass);
            if (3 <= MaxCharge) addIons_k<3>(spec, len_spec, b_mass, y_mass);
            if (4 <= MaxCharge) addIons_k<4>(spec, len_spec, b_mass, y_mass);
        }
    }
}

/*
 * Select a number of threads and blocks. Each block will have at least one full
 * warp, as required by the core kernel
 */
static void
addIons_control(uint32_t N, uint32_t &blocks, uint32_t &threads)
{
    threads = (N < MAX_THREADS) ? max(WARP_SIZE, ceilPow2(N)) : MAX_THREADS;
    blocks  = (N + threads - 1) / threads;
    blocks  = min(blocks, MAX_BLOCKS);
}


template <uint32_t MaxCharge>
static void
addIons_dispatch
(
    uint32_t            *d_spec,
    const float         *d_residual,
    const float         *d_ladder,
    const uint32_t      *d_rowPtr,
    const uint32_t      *d_inRangeIdx,
    const uint32_t      num_inRange,
    const uint32_t      len_spec
)
{
    uint32_t blocks;
    uint32_t threads;

    addIons_control(num_inRange, blocks, threads);
    switch (threads)
    {
    case 512: addIons_core<512,MaxCharge><<<blocks,threads>>>(d_spec, d_residual, d_ladder, d_rowPtr, d_inRangeIdx, num_inRange, len_spec); break;
    case 256: addIons_core<256,MaxCharge><<<blocks,threads>>>(d_spec, d_residual, d_ladder, d_rowPtr, d_inRangeIdx, num_inRange, len_spec); break;
    case 128: addIons_core<128,MaxCharge><<<blocks,threads>>>(d_spec, d_residual, d_ladder, d_rowPtr, d_inRangeIdx, num_inRange, len_spec); break;
    case  64: addIons_core< 64,MaxCharge><<<blocks,threads>>>(d_spec, d_residual, d_ladder, d_rowPtr, d_inRangeIdx, num_inRange, len_spec); break;
    case  32: addIons_core< 32,MaxCharge><<<blocks,threads>>>(d_spec, d_residual, d_ladder, d_rowPtr, d_inRangeIdx, num_inRange, len_spec); break;
    default:
        assert(!"Non-exhaustive patterns in match");
    }
}


void
addIons
(
    uint32_t            *d_spec,
    const float         *d_residual,
    const float         *d_ladder,
    const uint32_t      *d_rowPtr,
    const uint32_t      *d_inRangeIdx,
    const uint32_t      num_inRange,
    const uint32_t      max_charge,
    const uint32_t      len_spec
)
{
    switch (max_charge)
    {
    case 1: addIons_dispatch<1>(d_spec, d_residual, d_ladder, d_rowPtr, d_inRangeIdx, num_inRange, len_spec); break;
    case 2: addIons_dispatch<2>(d_spec, d_residual, d_ladder, d_rowPtr, d_inRangeIdx, num_inRange, len_spec); break;
    case 3: addIons_dispatch<3>(d_spec, d_residual, d_ladder, d_rowPtr, d_inRangeIdx, num_inRange, len_spec); break;
    case 4: addIons_dispatch<4>(d_spec, d_residual, d_ladder, d_rowPtr, d_inRangeIdx, num_inRange, len_spec); break;
    default:
        assert(!"Non-exhaustive patterns in match");
    }
}


