#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------------------
 *
 * Module    : Enum
 * Copyright : (c) 2009 Trevor L. McDonell
 * License   : BSD
 *
 * ---------------------------------------------------------------------------*/

#include "enum.h"

#include "algorithms.h"
#include "utils.h"
#include <stdint.h>


static void
enum_control(int32_t n, uint32_t &blocks, uint32_t &threads)
{
    threads = min(ceilPow2(n), MAX_THREADS);
    blocks  = (n + threads - 1) / threads;
}


template <typename T, bool increasing>
__global__ static void
enum_core(T *out, const T from, const T then, const T to)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const T val  = from + (then-from) * idx;

    if (increasing) if (val <= to) out[idx] = val;
    else            if (val >= to) out[idx] = val;
}

template <typename T>
static void
enumFromThenTo(T *out, const T from, const T then, const T to)
{
    uint32_t threads;
    uint32_t blocks;
    int32_t  n          = 1 + (to - from) / (then - from);
    bool     increasing = then-from > 0;

    if (n <= 0) return;

    enum_control(n, blocks, threads);
    if (increasing) enum_core<T,true> <<<blocks,threads>>>(out, from, then, to);
    else            enum_core<T,false><<<blocks,threads>>>(out, from, then, to);
}


// -----------------------------------------------------------------------------
// Instances
// -----------------------------------------------------------------------------

void enumFromTo_i(int32_t *out, int32_t l, int32_t u)
{
    enumFromThenTo<int32_t>(out, l, l+1, u);
}

