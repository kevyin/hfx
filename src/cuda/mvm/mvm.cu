/* -----------------------------------------------------------------------------
 *
 * Module    : MVM
 * Copyright : (c) [2012] Kevin Ying
 * License   : BSD
 *
 * Matrix vector multiplication using cublas
 * 
 * ---------------------------------------------------------------------------*/

#include "algorithms.h"

#include <stdint.h>
#include <thrust/device_vector.h>
#include "hipblas.h"

/* -----------------------------------------------------------------------------
 * Instances
 * ---------------------------------------------------------------------------*/

void
mvm_ff(hipblasHandle_t handle, float *d_y, const float *d_A, const float *d_x, const uint32_t m, const uint32_t n)
{
#ifdef _BENCH
    hipDeviceSynchronize();
    time_t t_beg, t_end;
    time(&t_beg);
    std::cerr << "mvm_ff" << std::endl;
#endif
    /*hipblasHandle_t handle;*/
    /*hipblasCreate(&handle);*/
    float alpha = 1.0;
    float beta = 0.0;
    // Because cublas uses col major storage (as opposed to row major) swap row and col values and use HIPBLAS_OP_T 
    /*thrust::device_ptr<const float> d_A_th(d_A);*/
    /*for (int i = 0; i < m; i++) {*/
        /*for (int j = 0; j < n; j++) {*/
            /*std::cerr << d_A_th[i*n + j] << " ";*/
        /*}*/
        /*std::cerr << std::endl;*/
    /*}*/
    //std::cerr << "m " << m << "n " << n << "alpha " << alpha << "beta " << beta << std::endl;
    hipblasStatus_t status = hipblasSgemv(handle, HIPBLAS_OP_T, n, m, &alpha, d_A, n, d_x, 1, &beta, d_y, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "CUBLAS FAILURE" << std::endl;
    }

    /*hipblasDestroy(handle);*/

#ifdef _BENCH
    hipDeviceSynchronize();
    time(&t_end);
    std::cerr<< "Time elapsed for mvm_ff: " << difftime(t_end,t_beg) << " seconds" << std::endl;
#endif

}
