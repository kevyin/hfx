#include "hip/hip_runtime.h"
/*
 * Module    : IonSeries
 * Copyright : (c) 2009 Trevor L. McDonell
 * License   : BSD
 */

#include "mass.h"
#include "utils.h"
#include "kernels.h"


/*
 * Convert a given mass into a mass/charge ratio
 * Locate the appropriate spectrum bin for a peak.
 */
__device__ float ionMZ(float m, float c) { return (m + massH * c) / c; }
__device__ int   bin(float x) { return rintf(x / binWidthMono); }


/*
 * Add a spectral peak for each fragment ion location, as well as the peaks
 * corresponding to the neutral losses of H2O and NH3.
 */
__device__ void
addIonsAB(float mass, float charge, int *spec, unsigned int N)
{
    int   idx;
    float m;

    // A
    idx = bin(ionMZ(mass - massCO, charge));
    if (0 <= idx && idx < N) atomicMax(&spec[idx], 10);

    // B
    m   = ionMZ(mass, charge);
    idx = bin(m);

    if (1 <= idx && idx < N-1)
    {
        atomicMax(&spec[idx],   50);
        atomicMax(&spec[idx+1], 25);
        atomicMax(&spec[idx-1], 25);
    }

    idx = bin(m - massH2O/charge);
    if (0 <= idx && idx < N) atomicMax(&spec[idx], 10);

    idx = bin(m - massNH3/charge);
    if (0 <= idx && idx < N) atomicMax(&spec[idx], 10);
}


__device__ void
addIonsY(float mass, float charge, int *spec, unsigned int N)
{
    float m   = ionMZ(mass + massH2O, charge);
    int   idx = bin(m);

    if (1 <= idx && idx < N-1)
    {
        atomicMax(&spec[idx],   50);
        atomicMax(&spec[idx+1], 25);
        atomicMax(&spec[idx-1], 25);
    }

    idx = bin(m - massNH3/charge);
    if (0 <= idx && idx < N) atomicMax(&spec[idx], 10);
}


/*
 * Add a spectral peak for each fragment ion location. The output spectrum array
 * must exist and be initialised to zero.
 */
__global__ static void
addIons_core
(
    int          max_charge,
    float        residual,
    float        *ladder,
    int          *spec,
    unsigned int len_ions,
    unsigned int len_spec
)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx < len_ions)
    {
        float y_mass = ladder[idx];
        float b_mass = residual - y_mass;

        for (int charge = 1; charge <= max_charge; ++charge)
        {
            addIonsAB(b_mass, (float) charge, spec, len_spec);
            addIonsY (y_mass, (float) charge, spec, len_spec);
        }
    }
}


void
addIons
(
    int                 max_charge,
    float               residual,
    float               *y_ions,
    int                 *spec,
    unsigned int        len_ions,
    unsigned int        len_spec,
    unsigned int        offset
)
{
    unsigned int threads = min(ceilPow2(len_ions), 512);
    unsigned int blocks  = (len_ions + threads - 1) / threads;

    /*
     * y_ions[offset] == residual
     */
    addIons_core<<<blocks,threads>>>(max_charge, residual, &y_ions[offset+1], spec, len_ions, len_spec);
}

