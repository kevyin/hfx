#include "hip/hip_runtime.h"
/*
 * Module    : IonSeries
 * Copyright : (c) 2009 Trevor L. McDonell
 * License   : BSD
 */

#include "mass.h"
#include "utils.h"
#include "kernels.h"


/*
 * Convert a given mass into a mass/charge ratio
 * Locate the appropriate spectrum bin for a peak.
 */
__device__ float ionMZ(float m, float c) { return (m + massH * c) / c; }
__device__ int   bin(float x) { return rintf(x / binWidthMono); }


/*
 * Add a spectral peak for each fragment ion location, as well as the peaks
 * corresponding to the neutral losses of H2O and NH3.
 */
__device__ void
addIonsAB(float mass, float charge, int *spec)
{
    // A
    atomicMax(&spec[bin(ionMZ(mass - massCO, charge))], 10);

    // B
    float m = ionMZ(mass, charge);
    int   b = bin(m);

    atomicMax(&spec[b],   50);
    atomicMax(&spec[b+1], 25);
    atomicMax(&spec[b-1], 25);
    atomicMax(&spec[bin(m - massH2O/charge)], 10);
    atomicMax(&spec[bin(m - massNH3/charge)], 10);
}


__device__ void
addIonsY(float mass, float charge, int *spec)
{
    float m = ionMZ(mass + massH2O, charge);
    int   b = bin(m);

    atomicMax(&spec[b],   50);
    atomicMax(&spec[b+1], 25);
    atomicMax(&spec[b-1], 25);
    atomicMax(&spec[bin(m - massNH3/charge)], 10);
}


/*
 * Add a spectral peak for each fragment ion location. The output spectrum array
 * must exist and be initialised to zero.
 */
template <bool lengthIsPow2>
__global__ static void
addIons_core
(
    int          max_charge,
    float        *b_ions,
    float        *y_ions,
    int          *spec,
    unsigned int len_ions
)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (lengthIsPow2 || idx < len_ions)
    {
        int   charge = 1;
        float b_mass = b_ions[idx];
        float y_mass = y_ions[idx];

        do
        {
            addIonsAB(b_mass, (float) charge, spec);
            addIonsY (y_mass, (float) charge, spec);
        }
        while (++charge < max_charge);
    }
}


void
addIons
(
    int          max_charge,
    float        *b_ions,
    float        *y_ions,
    int          *spec,
    unsigned int len_ions,
    unsigned int len_spec
)
{
    unsigned int threads = min(ceilPow2(len_ions), 512);
    unsigned int blocks  = (len_ions + threads - 1) / threads;

    (void) len_spec;

    if (isPow2(len_ions))
        addIons_core<true><<<blocks,threads>>>(max_charge, b_ions, y_ions, spec, len_ions);
    else
        addIons_core<false><<<blocks,threads>>>(max_charge, b_ions, y_ions, spec, len_ions);
}

